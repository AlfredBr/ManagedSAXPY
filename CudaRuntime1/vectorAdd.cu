#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"
{
    // CUDA device code
    __global__ void VectorAdd(const int* A, const int* B, int* C, int N)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N)
        {
            C[i] = A[i] + B[i];
        }
    }
}