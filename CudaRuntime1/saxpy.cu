#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"
{
    // CUDA device code
    __global__ void Saxpy(int N, float A, float* S, const float* X, const float* Y)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N)
        {
            S[i] = A * X[i] + Y[i];
        }
    }
}