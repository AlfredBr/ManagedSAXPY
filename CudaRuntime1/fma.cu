#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C"
{
    // CUDA device code
    __global__ void Fma(int N, float A, float* S, const float* X, const float* Y)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N)
        {
            S[i] = __fmaf_rn(A, X[i], Y[i]);
        }
    }
}